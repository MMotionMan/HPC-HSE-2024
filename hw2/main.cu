#include <iostream>
#include <cstdlib>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCK_SIZE 16
#define BLOCK_SIZE_WITH_PADDING (BLOCK_SIZE + 1)
const int N = 1024;

void generateMatrixVector(std::vector<float>& mat, int size) {
    for (int i = 0; i < size * size; ++i)
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
}

void generateMatrixArray(float* mat, int size) {
    for (int i = 0; i < size * size; ++i) {
	mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

__global__ void matrixMulKernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

__global__ void matrixMulKernelAsync(const float* A, const float* B, float* C, int N, int blockSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < blockSize && col < blockSize) {
	int globalRow = blockIdx.z * blockSize + row;
	int globalCol = blockIdx.z * blockSize + col;

	if (globalRow < N && globalCol < N) {
	    float sum = 0.0f;
	    for (int k = 0; k < N; ++k) {
		sum += A[globalRow * N + k] * B[k * N + globalCol];
	    }
	    C[globalRow * N + globalCol] = sum;
	}
    }
}

void matrixMultiplyOpenMP(float* A, float* B, float* C, int n) {
    #pragma omp target data map(to: A[0:n*n], B[0:n*n]) map(from: C[0:n*n])
    {
        #pragma omp target teams distribute parallel for collapse(2)
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < n; j++) {
                float sum = 0.0f;
                for (int k = 0; k < n; k++) {
                    sum += A[i * n + k] * B[k * n + j];
                }
                C[i * n + j] = sum;
            }
        }
    }
}

__global__ void matrixMulSharedKernel(const float* A, const float* B, float* C, int N) {
    __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    for (int blockIdx = 0; blockIdx < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; ++blockIdx)  {
	if (row < N && (blockIdx * BLOCK_SIZE + threadIdx.x) < N) {
	    sharedA[threadIdx.y][threadIdx.x] = A[row * N + blockIdx * BLOCK_SIZE + threadIdx.x];
	} else {
	    sharedA[threadIdx.y][threadIdx.x] = 0.0f;
	}

	if (col < N && (blockIdx * BLOCK_SIZE + threadIdx.y) < N) {
	    sharedB[threadIdx.y][threadIdx.x] = B[(blockIdx * BLOCK_SIZE + threadIdx.y) * N + col];
	} else {
	    sharedB[threadIdx.y][threadIdx.x] = 0.0f;
	}

	__syncthreads();

	for (int k = 0; k < BLOCK_SIZE; ++k) {
	    sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
	}

	__syncthreads();

	if (row < N && col < N) {
	    C[row * N +col] = sum;
	}
    }
}

__global__ void matrixMulSharedKernelWithPadding(const float* A, const float* B, float* C, int N) {
    __shared__ float sharedA[BLOCK_SIZE_WITH_PADDING][BLOCK_SIZE_WITH_PADDING];
    __shared__ float sharedB[BLOCK_SIZE_WITH_PADDING][BLOCK_SIZE_WITH_PADDING];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    for (int blockIdx = 0; blockIdx < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; ++blockIdx)  {
        if (row < N && (blockIdx * BLOCK_SIZE + threadIdx.x) < N) {
            sharedA[threadIdx.y][threadIdx.x] = A[row * N + blockIdx * BLOCK_SIZE + threadIdx.x];
        } else {
            sharedA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && (blockIdx * BLOCK_SIZE + threadIdx.y) < N) {
            sharedB[threadIdx.y][threadIdx.x] = B[(blockIdx * BLOCK_SIZE + threadIdx.y) * N + col];
        } else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }

        __syncthreads();

        if (row < N && col < N) {
            C[row * N +col] = sum;
        }
    }
}

void measureExecutionTime(float* d_A, float* d_B, float* d_C, int N, bool usePadding) {
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipMemset(d_C, 0, N * N * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    if (usePadding) {
        matrixMulSharedKernelWithPadding<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    } else {
	matrixMulSharedKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "Use padding = " << usePadding << "time = " << milliseconds << std::endl;
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
	std::cerr << "CUDA ERROR: " << msg << " - " << hipGetErrorString(err) << std::endl;
	exit(EXIT_FAILURE);
    }
}

void baseRealization() {
    const int size = N * N;
    const size_t bytes = size * sizeof(float);

    std::vector<float> h_A(size), h_B(size), h_C(size);

    generateMatrixVector(h_A, N);
    generateMatrixVector(h_B, N);

    float *d_A, *d_B, *d_C;

    checkCudaError(hipMalloc(&d_A, bytes), "Allocating d_A");
    checkCudaError(hipMalloc(&d_B, bytes), "Allocating d_B");
    checkCudaError(hipMalloc(&d_C, bytes), "Allocating d_C");

    checkCudaError(hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice), "Copying A to d_A");
    checkCudaError(hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice), "Copying B to d_B");

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    checkCudaError(hipGetLastError(), "Kernel execution");

    checkCudaError(hipMemcpy(h_C.data(), d_C, bytes, hipMemcpyDeviceToHost), "Copying C to host");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << "C[0][0] = " << h_C[0] << std::endl;
}

void pinnedMemoryRealization() {
    const int N = 1024;
    size_t bytes = N * N * sizeof(float);

    float *h_A, *h_B, *h_C;
    hipHostAlloc(&h_A, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_B, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_C, bytes, hipHostMallocDefault);
    
    srand(time(NULL));

    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    matrixMulKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    float sum = 0.0f;
    for (int i = 0; i < N * N; i++) {
        sum += h_C[i];
    }
    printf("Сумма элементов результирующей матрицы: %f\n", sum);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void unifiedMemoryRealization() {
    const int N = 1024; // Размер матриц NxN
    size_t bytes = N * N * sizeof(float);

    srand(time(NULL));

    float *A, *B, *C;
    hipMallocManaged(&A, bytes);
    hipMallocManaged(&B, bytes);
    hipMallocManaged(&C, bytes);

    generateMatrixArray(A, N);
    generateMatrixArray(B, N);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    matrixMulKernel<<<gridSize, blockSize>>>(A, B, C, N);

    hipDeviceSynchronize();

    float sum = 0.0f;
    for (int i = 0; i < N * N; ++i) {
        sum += C[i];
    }
    printf("Сумма всех элементов результирующей матрицы: %f\n", sum);

    hipFree(A);
    hipFree(B);
    hipFree(C);
}

void cudaStreamsRealization() {
    const int N = 1024;    
    const int blockSize = 256;
    const int numStreams = 4;
    size_t bytes = N * N * sizeof(float);

    srand(time(NULL));

    float *h_A, *h_B, *h_C;
    hipHostAlloc(&h_A, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_B, bytes, hipHostMallocDefault);
    hipHostAlloc(&h_C, bytes, hipHostMallocDefault);

    generateMatrixArray(h_A, N);
    generateMatrixArray(h_B, N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    dim3 blockDim(16, 16);
    dim3 gridDim((blockSize + blockDim.x - 1) / blockDim.x, (blockSize + blockDim.y - 1) / blockDim.y, 1);

    for (int streamIdx = 0; streamIdx < numStreams; ++streamIdx) {
        int offset = streamIdx * blockSize * blockSize;
        size_t blockBytes = blockSize * blockSize * sizeof(float);

        hipMemcpyAsync(&d_A[offset], &h_A[offset], blockBytes, hipMemcpyHostToDevice, streams[streamIdx]);

        matrixMulKernelAsync<<<gridDim, blockDim, 0, streams[streamIdx]>>>(d_A, d_B, d_C, N, blockSize);

        hipMemcpyAsync(&h_C[offset], &d_C[offset], blockBytes, hipMemcpyDeviceToHost, streams[streamIdx]);
    }

    for (int i = 0; i < numStreams; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    float sum = 0.0f;
    for (int i = 0; i < N * N; ++i) {
        sum += h_C[i];
    }
    printf("Сумма всех элементов результирующей матрицы: %f\n", sum);
    
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    for (int i = 0; i < numStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }
}

void sharedMemoryRealization() {
    const int N = 1024;
    size_t bytes = N * N * sizeof(float);

    srand(time(NULL));

    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);

    generateMatrixArray(h_A, N);
    generateMatrixArray(h_B, N);
    
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    matrixMulSharedKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int i = 0; i < N * N; ++i) {
        sum += h_C[i];
    }
    printf("Сумма всех элементов результирующей матрицы: %f\n", sum);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void sharedMemoryOptimRealization() {
    const int N = 1024;
    size_t bytes = N * N * sizeof(float);
    srand(time(NULL));
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);

    generateMatrixArray(h_A, N);
    generateMatrixArray(h_B, N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    measureExecutionTime(d_A, d_B, d_C, N, true);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);

    generateMatrixArray(h_A, N);
    generateMatrixArray(h_B, N);
    
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes); 
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    measureExecutionTime(d_A, d_B, d_C, N, false);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);    
}

void mulWithCuBLASS(){
    int bytes = N * N * sizeof(float);

    float* h_A = new float[N * N];
    float* h_B = new float[N * N];
    float* h_C = new float[N * N];

    generateMatrixArray(h_A, N);
    generateMatrixArray(h_B, N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, N, N,
                &alpha,
                d_A, N,
                d_B, N,
                &beta,
                d_C, N);
    
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    float sum = 0.0f;
    for (int i = 0; i < N * N; ++i) {
        sum += h_C[i];
    }
    std::cout << "Сумма всех элементов результирующей матрицы: " << sum << std::endl;    

    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int main() {
    baseRealization();
    pinnedMemoryRealization();
    unifiedMemoryRealization();
    cudaStreamsRealization();
    sharedMemoryRealization();
    sharedMemoryOptimRealization();
    mulWithCuBLASS();
    return 0;
}
